#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100000

__global__ void vectorAdd(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    int size = N * sizeof(int);

    // Initialize input vectors a and b
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    // Copy input vectors from host to device memory
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    // Launch the vectorAdd kernel on the device
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c);

    // Copy the result vector from device to host memory
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; i++) {
        if (c[i] != a[i] + b[i]) {
            printf("Error: c[%d] != a[%d] + b[%d]\n", i, i, i);
            break;
        }
    }

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
